#include <iostream>
#include <hip/hip_runtime.h>
#include <cmath>
#include <vector>
#include <cstdlib>
#include <algorithm>
#include <iterator>

#define CUDA_CALL(x)                                                                                                            \
    {                                                                                                                           \
        const hipError_t a = (x);                                                                                              \
        if (a != hipSuccess)                                                                                                   \
        {                                                                                                                       \
            std::cerr << "CUDA Runtime Error: " << hipGetErrorString(a) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
            exit(EXIT_FAILURE);                                                                                                 \
        }                                                                                                                       \
    }

// Network parameters
const int input_size = 27;
const int hidden_size = 128;
const int output_size = 27;
const float learning_rate = 0.0001;

// Activation function
__device__ float sigmoid(float x)
{
    return 1.0f / (1.0f + exp(-x));
}

__device__ float sigmoid_derivative(float x)
{
    float s = sigmoid(x);
    return s * (1.0f - s);
}

// Forward pass kernel
__global__ void forward_pass(float *x, float *y_pred, float *d_input_hidden_weights, float *d_hidden_output_weights, float *d_hidden_bias, float *d_output_bias, float *hidden_layer_output)
{
    int tid = threadIdx.x;

    // Calculate hidden layer activations
    float h_value = 0.0;
    for (int i = 0; i < input_size; i++)
    {
        h_value += d_input_hidden_weights[tid * input_size + i] * x[i];
    }
    h_value += d_hidden_bias[tid];
    hidden_layer_output[tid] = sigmoid(h_value);

    // Calculate output layer activations (one thread calculates one output neuron's value)
    float o_value = 0.0;
    for (int j = 0; j < hidden_size; j++)
    {
        o_value += d_hidden_output_weights[tid * hidden_size + j] * hidden_layer_output[j];
    }
    o_value += d_output_bias[tid];
    y_pred[tid] = sigmoid(o_value);
}

// Backward pass kernel
__global__ void backward_pass(float *x, float *y, float *y_pred, float *d_input_hidden_weights, float *d_hidden_output_weights, float *d_hidden_bias, float *d_output_bias, float *hidden_layer_output)
{
    int tid = threadIdx.x;

    // Calculate error at the output
    float delta_output = (y[tid] - y_pred[tid]) * sigmoid_derivative(y_pred[tid]);

    // Update output weights and bias
    for (int j = 0; j < hidden_size; j++)
    {
        d_hidden_output_weights[tid * hidden_size + j] += learning_rate * delta_output * hidden_layer_output[j];
    }
    d_output_bias[tid] += learning_rate * delta_output;

    // Calculate error for hidden layer
    float delta_hidden = 0.0;
    for (int k = 0; k < output_size; k++)
    {
        delta_hidden += d_hidden_output_weights[k * hidden_size + tid] * delta_output;
    }

    delta_hidden *= sigmoid_derivative(hidden_layer_output[tid]);

    // Update input weights and bias
    for (int i = 0; i < input_size; i++)
    {
        d_input_hidden_weights[tid * input_size + i] += learning_rate * delta_hidden * x[i];
    }
    d_hidden_bias[tid] += learning_rate * delta_hidden;
}

void train(float *x, float *y, float *d_input_hidden_weights, float *d_hidden_output_weights, float *d_hidden_bias, float *d_output_bias)
{
    float *d_x, *d_y, *y_pred, *hidden_layer_output;

    CUDA_CALL(hipMalloc(&d_x, input_size * sizeof(float)));
    CUDA_CALL(hipMalloc(&d_y, output_size * sizeof(float)));
    CUDA_CALL(hipMalloc(&y_pred, output_size * sizeof(float)));
    CUDA_CALL(hipMalloc(&hidden_layer_output, hidden_size * sizeof(float)));

    CUDA_CALL(hipMemcpy(d_x, x, input_size * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_y, y, output_size * sizeof(float), hipMemcpyHostToDevice));

    for (int epoch = 0; epoch < 1000; epoch++)
    { // Assuming 1000 epochs for demonstration
        forward_pass<<<1, hidden_size>>>(d_x, y_pred, d_input_hidden_weights, d_hidden_output_weights, d_hidden_bias, d_output_bias, hidden_layer_output);
        backward_pass<<<1, hidden_size>>>(d_x, d_y, y_pred, d_input_hidden_weights, d_hidden_output_weights, d_hidden_bias, d_output_bias, hidden_layer_output);
    }

    hipFree(d_x);
    hipFree(d_y);
    hipFree(y_pred);
    hipFree(hidden_layer_output);
}

void printWeightsAndBiases(float *d_weights, int rows, int cols, const char *name)
{
    std::vector<float> h_weights(rows * cols);

    // Copy from device to host
    CUDA_CALL(hipMemcpy(h_weights.data(), d_weights, rows * cols * sizeof(float), hipMemcpyDeviceToHost));

    std::cout << name << ":" << std::endl;
    for (int i = 0; i < rows; ++i)
    {
        for (int j = 0; j < cols; ++j)
        {
            std::cout << h_weights[i * cols + j] << " ";
        }
        std::cout << std::endl;
    }
}

void printBias(float *h_bias, int size, const char *name)
{
    std::cout << name << ":" << std::endl;
    for (int i = 0; i < size; ++i)
    {
        std::cout << h_bias[i] << " ";
    }
    std::cout << std::endl;
}

std::vector<float> charToOneHot(char c)
{
    std::vector<float> oneHot(input_size, 0.0f);
    int index = c - 'a';
    if (index >= 0 && index < input_size)
    {
        oneHot[index] = 1.0f;
    }
    else
    {
        oneHot[input_size - 1] = 1.0f; // Let's use the last position for any non a-z character
    }
    return oneHot;
}

std::vector<std::vector<float>> stringToInputData(const std::string &s)
{
    std::vector<std::vector<float>> data;
    for (char c : s)
    {
        data.push_back(charToOneHot(c));
    }
    return data;
}

void initializeWeightsAndBiases(float **d_weights, int size)
{
    std::vector<float> h_weights(size);
    for (auto &w : h_weights)
    {
        w = (rand() / float(RAND_MAX)) * sqrt(2.0 / size);
    }
    CUDA_CALL(hipMalloc((void **)d_weights, size * sizeof(float)));
    hipMemcpy(*d_weights, h_weights.data(), size * sizeof(float), hipMemcpyHostToDevice);
}

std::string predict(const std::string &input_str, float *d_input_hidden_weights, float *d_hidden_output_weights, float *d_hidden_bias, float *d_output_bias)
{
    auto inputData = stringToInputData(input_str);
    float *d_x, *y_pred, *hidden_layer_output;
    std::vector<float> h_output(output_size);

    CUDA_CALL(hipMalloc(&d_x, input_size * sizeof(float)));
    CUDA_CALL(hipMalloc(&y_pred, output_size * sizeof(float)));
    CUDA_CALL(hipMalloc(&hidden_layer_output, hidden_size * sizeof(float)));

    hipMemcpy(d_x, inputData.back().data(), input_size * sizeof(float), hipMemcpyHostToDevice);

    forward_pass<<<1, hidden_size>>>(d_x, y_pred, d_input_hidden_weights, d_hidden_output_weights, d_hidden_bias, d_output_bias, hidden_layer_output);
    // CUDA_CALL(cudaPeekAtLastError());
    CUDA_CALL(hipDeviceSynchronize());

    CUDA_CALL(hipMemcpy(h_output.data(), y_pred, output_size * sizeof(float), hipMemcpyDeviceToHost));

    int max_idx = std::distance(h_output.begin(), std::max_element(h_output.begin(), h_output.end()));
    char predicted_char = (max_idx == input_size - 1) ? ' ' : 'a' + max_idx;

    hipFree(d_x);
    hipFree(y_pred);
    hipFree(hidden_layer_output);

    return std::string(1, predicted_char);
}

int main()
{
    float *d_input_hidden_weights, *d_hidden_output_weights, *d_hidden_bias, *d_output_bias;

    initializeWeightsAndBiases(&d_input_hidden_weights, input_size * hidden_size);
    initializeWeightsAndBiases(&d_hidden_output_weights, hidden_size * output_size);
    initializeWeightsAndBiases(&d_hidden_bias, hidden_size);
    initializeWeightsAndBiases(&d_output_bias, output_size);

    std::string training_data = "hello, lorem ipsum dolor sit amet consectetur adipiscing elit, sed do eiusmod tempor incididunt ut labore et dolore magna aliqua. Ut enim ad minim veniam, quis nostrud exercitation ullamco laboris nisi ut aliquip ex ea commodo consequat.";
    auto inputData = stringToInputData(training_data);

    printWeightsAndBiases(d_hidden_output_weights, hidden_size, output_size, "Hidden to Output Weights");
    for (int i = 0; i < inputData.size() - 1; i++)
    {
        train(inputData[i].data(), inputData[i + 1].data(), d_input_hidden_weights, d_hidden_output_weights, d_hidden_bias, d_output_bias);
    }
    // After training
    std::cout << "\nAfter Training:" << std::endl;
    CUDA_CALL(hipDeviceSynchronize());
    printWeightsAndBiases(d_hidden_output_weights, hidden_size, output_size, "Hidden to Output Weights");

    // User interaction loop
    while (true)
    {
        std::string input_str;
        std::cout << "Enter string (or 'exit' to quit): ";
        std::getline(std::cin, input_str);
        if (input_str == "exit")
            break;
        if (input_str == "")
            continue;

        std::string predicted_str = predict(input_str, d_input_hidden_weights, d_hidden_output_weights, d_hidden_bias, d_output_bias);
        std::cout << "Predicted next character: " << predicted_str << std::endl;
    }

    CUDA_CALL(hipFree(d_input_hidden_weights));
    CUDA_CALL(hipFree(d_hidden_output_weights));
    CUDA_CALL(hipFree(d_hidden_bias));
    CUDA_CALL(hipFree(d_output_bias));

    return 0;
}